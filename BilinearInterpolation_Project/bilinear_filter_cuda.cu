#include <cstdlib>

#include "bilinear_filter_cuda.cuh"

#include "cuda_includes.h"
#include "common_structs.cuh"
#include "cuda_kernels.cuh"

namespace cuda_seq
{

	void create_pixel_precalculation(pixel_precalculation_memory* precalculation_xy, unsigned int old_width, unsigned int new_width, const unsigned old_height, const unsigned new_height);

	void fill_image_to_scale(png_user_struct* image_to_scale, png_user_struct* source_image, pixel_precalculation* d_x_pixel_precalculation_ptr,
		pixel_precalculation* d_y_pixel_precalculation_ptr);

	void scale_bilinear(png_user_struct* source_image, png_user_struct* image_to_scale)
	{
		const auto old_height = source_image->image_info.height;
		const auto new_height = image_to_scale->image_info.height;
		const auto old_width = source_image->image_info.width;
		const auto new_width = image_to_scale->image_info.width;

		pixel_precalculation_memory precalculation_xy;

		create_pixel_precalculation(&precalculation_xy, old_width, new_width, old_height, new_height);

		fill_image_to_scale(image_to_scale, source_image, precalculation_xy.d_x, precalculation_xy.d_y);

		hipFree(precalculation_xy.allocated_gpu_memory);
	}

	void create_pixel_precalculation(pixel_precalculation_memory* precalculation_xy,
		const unsigned int old_width, const unsigned int new_width, const unsigned old_height, const unsigned new_height)
	{
		auto needed_memory_in_bytes = sizeof(pixel_precalculation) * new_width + sizeof(pixel_precalculation) * new_height;

		auto offset_to_y_precalculation_data = new_width;

		pixel_precalculation* d_memory_on_gpu_p;

		hipMalloc(reinterpret_cast<void**>(&d_memory_on_gpu_p), needed_memory_in_bytes);
		precalculation_xy->allocated_gpu_memory = d_memory_on_gpu_p;
		precalculation_xy->d_x = d_memory_on_gpu_p;
		precalculation_xy->d_y = d_memory_on_gpu_p + offset_to_y_precalculation_data;


		//old_size - 1, the last source pixel is (old_size - 1)
		const auto pixel_weight_increment_x = (1.0F / static_cast<float>(new_width)) * static_cast<float>(old_width - 1);
		const auto pixel_weight_increment_y = (1.0F / static_cast<float>(new_height)) * static_cast<float>(old_height - 1);

		dim3 blockSize_x(32);
		dim3 blockSize_y(32);

		auto bx = (new_width + blockSize_x.x - 1) / blockSize_x.x;
		auto by = (new_height + blockSize_y.x - 1) / blockSize_y.x;

		auto gridSize_x = dim3(bx);
		auto gridSize_y = dim3(by);

		pixel_precalculation_kernel << < gridSize_x, blockSize_x >> > (precalculation_xy->d_x, pixel_weight_increment_x, new_width);  // NOLINT(clang-diagnostic-unused-value)
		pixel_precalculation_kernel << < gridSize_y, blockSize_y >> > (precalculation_xy->d_y, pixel_weight_increment_y, new_height); // NOLINT(clang-diagnostic-unused-value)

	}

	void fill_image_to_scale(png_user_struct* image_to_scale, png_user_struct* source_image, pixel_precalculation* d_x_pixel_precalculation_ptr,
		pixel_precalculation* d_y_pixel_precalculation_ptr)
	{
		auto dimensions_size_in_bytes = sizeof(dimensions_info);
		auto source_png_size_in_bytes = sizeof(png_byte) * source_image->image_info.width * source_image->image_info.height;
		auto image_to_scale_size_in_bytes = sizeof(png_byte) * image_to_scale->image_info.width * image_to_scale->image_info.height;
		auto sobel_image_size_in_bytes = image_to_scale_size_in_bytes;

		auto needed_memory_in_bytes = dimensions_size_in_bytes + source_png_size_in_bytes + image_to_scale_size_in_bytes + sobel_image_size_in_bytes;

		_int8* allocated_memory_on_gpu_p;

		hipMalloc(reinterpret_cast<void**>(&allocated_memory_on_gpu_p), needed_memory_in_bytes);

		d_scale_params hd_params;
		d_sobel_params hd_sobel_params;

		hd_params.d_x_precalculation_p = d_x_pixel_precalculation_ptr;
		hd_params.d_y_precalculation_p = d_y_pixel_precalculation_ptr;
		hd_params.d_dimensions_info_p = reinterpret_cast<dimensions_info*>(allocated_memory_on_gpu_p);
		hd_params.d_source_bytes_sequential_p = reinterpret_cast<png_bytep>(allocated_memory_on_gpu_p + dimensions_size_in_bytes);
		hd_params.image_to_scale_bytes_sequential_p = reinterpret_cast<png_bytep>(allocated_memory_on_gpu_p + dimensions_size_in_bytes + source_png_size_in_bytes);
		hd_sobel_params.source_bytes_sequential_p = hd_params.image_to_scale_bytes_sequential_p;
		hd_sobel_params.result_bytes_sequential_p = reinterpret_cast<png_bytep>(allocated_memory_on_gpu_p + dimensions_size_in_bytes + source_png_size_in_bytes + image_to_scale_size_in_bytes);

		
		//pp_array that contains the source image needs to be flattened for fast memory allocation on gpu
		png_bytep png_source_bytes_p = png_util_create_flat_bytes_p_from_row_pp(source_image->png_rows, source_image->image_info.width, source_image->image_info.height, source_png_size_in_bytes);

		hipMemcpy(hd_params.d_source_bytes_sequential_p, png_source_bytes_p, source_png_size_in_bytes, hipMemcpyHostToDevice);
		
		dimensions_info dimensions_inf;
		dimensions_inf.image_to_scale_width = image_to_scale->image_info.width;
		dimensions_inf.image_to_scale_height = image_to_scale->image_info.height;
		dimensions_inf.source_image_width = source_image->image_info.width;

		hipMemcpy(hd_params.d_dimensions_info_p, &dimensions_inf, sizeof(dimensions_info), hipMemcpyHostToDevice);

		hd_sobel_params.dimensions_inf_p = hd_params.d_dimensions_info_p;

		dim3 blockSize(32, 32);

		auto bx = (image_to_scale->image_info.width + blockSize.x - 1) / blockSize.x;

		auto by = (image_to_scale->image_info.height + blockSize.y - 1) / blockSize.y;

		auto gridSize = dim3(bx, by);

		parallel_tasks_bilinear_nn_sobel << <gridSize, blockSize >> > (hd_params, hd_sobel_params);

		hipMemcpy(image_to_scale->png_rows[0], hd_sobel_params.result_bytes_sequential_p, image_to_scale_size_in_bytes, hipMemcpyDeviceToHost);

		hipFree(allocated_memory_on_gpu_p);
	}

}
